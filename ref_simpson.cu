#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#define warp_size 32
#define Hwarp_size 16
#define N_points 33554432
#define A 0
#define B 15

void checkCUDAError(const char* msg);

__host__ __device__ inline double f(double x)
{
	return exp(x)*sin(x);
}

__global__ void fn_evalCalc(double *fn_eval, double a, double b)	//N_points/numBlocks should be a integer.
{
	extern __shared__ double local_array[];
	double step = (b-a)/N_points, mult, sum = 0.0, diff = (b-a)/gridDim.x;
	int eval = N_points/gridDim.x;
	//b = a + (blockIdx.x+1)*diff;
	a += blockIdx.x*diff;
	
	for(int k = threadIdx.x; k < eval; k += blockDim.x)
	{
		mult = (k%2==0)?2.0:4.0;
		sum += mult*f(a + step*k);
	}
	local_array[threadIdx.x] = sum;
	__syncthreads();
	
	//BlockReduce.
	for(int s = 1; s < blockDim.x; s *= 2) 
	{
      if ((threadIdx.x % (2*s)) == 0) 
            local_array[threadIdx.x] += local_array[threadIdx.x + s];
      __syncthreads();
    }
    
    if(!threadIdx.x)
		fn_eval[blockIdx.x] = local_array[threadIdx.x];
	
}



__global__ void globalReduce(double *fn_eval, double a, double b, int size)
{
		extern __shared__ double local_array[];
		double step = (b-a)/N_points;
		if(threadIdx.x < size)
			local_array[threadIdx.x] = fn_eval[threadIdx.x];

		
		for(int s = 1; s < blockDim.x; s *= 2) 
		{
			if ((threadIdx.x % (2*s)) == 0) 
				local_array[threadIdx.x] += local_array[threadIdx.x + s];
			__syncthreads();
		}
		if(!threadIdx.x)
			fn_eval[0] = step*(local_array[threadIdx.x] + f(b) - f(a))/3;
}


int main( int argc, char** argv)
{
	double sum=0.0,*d_fn_eval;
	int numBlocks = 128, numThreadsPerBlock = 64; //keep numBlocks within 1024
	hipDeviceSetCacheConfig(hipFuncCachePreferShared);
	
	timeval t;
	double t1,t2;
	
	hipMalloc( (void **) &d_fn_eval, sizeof(double) );
	gettimeofday(&t, NULL);
	t1 = t.tv_sec*1000.0 + (t.tv_usec/1000.0);
	
	
	fn_evalCalc<<< numBlocks, numThreadsPerBlock, numThreadsPerBlock*sizeof(double) >>>(d_fn_eval,A,B);
	globalReduce<<< 1, numBlocks, numBlocks*sizeof(double) >>>(d_fn_eval,A,B,numBlocks);
	hipDeviceSynchronize();
	
	gettimeofday(&t, NULL);
	t2 = t.tv_sec*1000.0 + (t.tv_usec/1000.0);
	checkCUDAError("kernel invocation");
	hipMemcpy( &sum, d_fn_eval, sizeof(double), hipMemcpyDeviceToHost );
	checkCUDAError("memcpy");

	//for(int k = 0; k<N_points; k++ )
	//	printf("%lf\t",h_fn_eval[k]);
	//for(int k=0;k<numBlocks;k++)
	//	sum+=h_debug_output[k];
	printf("%lf~~~TIME : %lf ms\n\n\n",sum,t2-t1);//,sum);
}

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }
}
