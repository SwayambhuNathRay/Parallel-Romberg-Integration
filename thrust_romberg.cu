#include "hip/hip_runtime.h"
#include <thrust/sequence.h>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <sys/time.h>


#define pi_f  3.14159265358979f                 // Greek pi in single precision

struct sin_functor
{
    __host__ __device__
    float operator()(float x) const
    {
        return x*sin(x);
    }
};

int main(void)
{
    int M = 12;                          // --- Maximum number of Romberg iterations

    float a     = 0.f;                  // --- Lower integration limit
    float b     = 1000.f;                  // --- Upper integration limit

    float hmin  = (b-a)/pow(2.f,M-1);   // --- Minimum integration step size 

    // --- Define the matrix for Romberg approximations and initialize to 1.f 
    
    timeval t;
	double t1,t2,t3,t4;
	
	gettimeofday(&t, NULL);
	t1 = t.tv_sec*1000.0 + (t.tv_usec/1000.0);
    thrust::host_vector<float> R(M*M,1.f);

    for (int k=0; k<M; k++) {

        float h = pow(2.f,k-1)*hmin;    // --- Step size for the k-th row of the Romberg matrix

        // --- Define integration nodes
        int N = (int)((b - a)/h) + 1;
        thrust::device_vector<float> d_x(N);
        thrust::sequence(d_x.begin(), d_x.end(), a, h);

        // --- Calculate function values
        thrust::device_vector<float> d_y(N);
        thrust::transform(d_x.begin(), d_x.end(), d_y.begin(), sin_functor());

        // --- Calculate integral
        R[k*M] = (.5f*h) * (d_y[0] + 2.f*thrust::reduce(d_y.begin() + 1, d_y.begin() + N - 1, 0.0f) + d_y[N-1]);

    }

    // --- Compute the k-th column of the Romberg matrix
    for (int k=1; k<M; k++) { 

        // --- The matrix of Romberg approximations is triangular!
        for (int kk=0; kk<(M-k+1); kk++) { 

            // --- See the Romberg integration algorithm
            R[kk*M+k] = R[kk*M+k-1] + (R[kk*M+k-1] - R[(kk+1)*M+k-1])/(pow(4.f,k)-1.f); 

        } 

    }
    
    gettimeofday(&t, NULL);
	t2 = t.tv_sec*1000.0 + (t.tv_usec/1000.0);

    // --- Define the vector Rnum for numerical approximations
    thrust::host_vector<float> Rnum(M); 
    thrust::copy(R.begin(), R.begin() + M, Rnum.begin());
    
   
	printf("TIME : %lf ms\n",t2-t1);

    for (int i=0; i<M; i++) printf("%i %f\n",i,Rnum[i]);

    //getchar();

    return 0;
}
