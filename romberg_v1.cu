#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#define warp_size 32
#define Hwarp_size 16
#define A 0
#define B 15

void checkCUDAError(const char* msg);

__host__ __device__ inline double f(double x)
{
	return exp(x)*sin(x);
}

__global__ void romberg(double a, double b, int max_eval, double *result)
{
	extern __shared__ double local_array[];
	double diff = (b-a)/gridDim.x, step;
	b = a + (blockIdx.x+1)*diff;
	a += blockIdx.x*diff;
	
	step = (b-a)/max_eval;
	for(int k = threadIdx.x; k < max_eval+1; k += blockDim.x)
		local_array[k] = f(a + step*k);
	
	//for(int k = threadIdx.x; k < max_eval+1; k += blockDim.x)
	//	result[blockIdx.x*(max_eval+1)+k] = local_array[k];

	if(threadIdx.x < 13)
	{
		int inc = 1<<(12-threadIdx.x);
		double sum = 0.0;
		for(int k = 0;k <= max_eval;k = k+inc)
		{
			sum += 2.0*local_array[k];
		}
		sum -= (local_array[0] + local_array[max_eval]);
		sum *= (b-a)/(1<<(threadIdx.x+1));
		local_array[threadIdx.x] = sum;
	}
	
	if(!threadIdx.x)
	{
		double romberg_table[13];
		for(int k=0;k<13;k++)
			romberg_table[k] = local_array[k];
		
		for(int col = 0 ; col < 12 ; col++)
		{
			for(int row = 12; row > col; row--)
			{
				romberg_table[row] = romberg_table[row] + (romberg_table[row] - romberg_table[row-1])/((1<<(2*col+1))-1);
			}
		}
		result[blockIdx.x] = romberg_table[12];
	}

}


int main( int argc, char** argv)
{
	double *d_result, *h_result,sum=0.0;
	int numBlocks = 128, numThreadsPerBlock = 64, max_eval = 4096;
	hipDeviceSetCacheConfig(hipFuncCachePreferShared);
    hipMalloc( (void **) &d_result, numBlocks*sizeof(double) );
	h_result = new double[numBlocks];
	
	timeval t;
	double t1,t2;
	
	gettimeofday(&t, NULL);
	t1 = t.tv_sec*1000.0 + (t.tv_usec/1000.0);
	
	
	romberg<<< numBlocks, numThreadsPerBlock, (max_eval+1)*sizeof(double) >>>(A,B,max_eval,d_result);
	hipDeviceSynchronize();
	
	gettimeofday(&t, NULL);
	t2 = t.tv_sec*1000.0 + (t.tv_usec/1000.0);
	checkCUDAError("kernel invocation");
	hipMemcpy( h_result, d_result, numBlocks*sizeof(double), hipMemcpyDeviceToHost );
	checkCUDAError("memcpy");
	
	//for(int k = 0; k<(max_eval+1)*numBlocks; k++ )
	//	printf("%lf\t",h_result[k]);
	for(int k=0;k<numBlocks;k++)
		sum+=h_result[k];
	printf("TIME : %lf ms with ans = %lf\n\n\n",t2-t1,sum);
}

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }
}
